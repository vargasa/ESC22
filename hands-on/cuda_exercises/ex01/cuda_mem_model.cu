#include "hip/hip_runtime.h"
// C++ standard headers
#include <cassert>
#include <iostream>
#include <vector>

// CUDA headers
#include <hip/hip_runtime.h>

// local headers
#include "cuda_check.h"

// Here you can set the device ID that was assigned to you
#define MYDEVICE 0

__global__ void sum(const float* a, const float* b, float* c){
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    c[tid] = a[tid] + b[tid];
}

///////////////////////////////////////////////////////////////////////////////
// Program main
///////////////////////////////////////////////////////////////////////////////
int main()
{
  // Choose one CUDA device
  CUDA_CHECK(hipSetDevice(MYDEVICE));

  // Create a CUDA stream to execute asynchronous operations on this device
  //hipStream_t stream;
  //CUDA_CHECK(hipStreamCreate(&stream));

  // Pointer and dimension for host memory
  size_t dimA = 1024*2;
  std::vector<float> h_a(dimA);
  std::vector<float> h_b(dimA);
  std::vector<float> h_c(dimA);

  // Allocate and initialize host memory
  for (uint i = 0; i < dimA; ++i) {
     h_a[i] = i;
     h_b[i] = 2*i;
  }

  // Pointers for device memory
  float *d_a, *d_b, *d_c;

  // Part 1 of 5: allocate the device memory
  size_t memSize = dimA * sizeof(float);
  //size_t floatSize = sizeof(float);

  hipMalloc(&d_a, memSize);
  hipMalloc(&d_b, memSize);
  hipMalloc(&d_c, memSize);

  // CUDA_CHECK(hipMallocAsync(___));
  // CUDA_CHECK(hipMallocAsync(___));

  // Part 2 of 5: host to device memory copy
  // Hint: the raw pointer to the underlying array of a vector
  // can be obtained by calling std::vector<T>::data()
  hipMemcpy(d_a, h_a.data(), memSize, hipMemcpyHostToDevice);
  hipMemcpy(d_b, h_b.data(), memSize, hipMemcpyHostToDevice);
  
  //CUDA_CHECK(hipMemcpyAsync(___));

  // Part 3 of 5: device to device memory copy
  //CUDA_CHECK(hipMemcpyAsync(___));
  // hipMemcpy(d_b, d_a, memSize, hipMemcpyDeviceToDevice);
  // hipMemcpy(d_a, d_b, memSize, hipMemcpyDeviceToDevice);
  
  // Clear the host memory
  //std::fill(h_a.begin(), h_a.end(), 0);

  // Part 4 of 5: device to host copy
  dim3 blocks(2,1,1); // define the number of blocks available as a 3d shape
  //dim3 blocks();
  //blocks.x = 1
  sum<<<blocks /*number of blocks*/, 1024 /*threads within the block*/, 0>>>(d_a,d_b,d_c);
  hipDeviceSynchronize();
  //hipStreamSynchronize(stream);
  hipMemcpy(h_c.data(),d_c, memSize, hipMemcpyDeviceToHost);

  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);

  for (int i = 0; i < dimA; ++i) {
    std::cout << h_c[i] << '\n';
    //assert(h_c[i] == 3.*i );
  }


  //CUDA_CHECK(hipMemcpyAsync(___));

  // Wait for all asynchronous operations to complete
  //CUDA_CHECK(hipStreamSynchronize(stream));

  // Part 5 of 5: free the device memory
  //CUDA_CHECK(hipFreeAsync(___));
  //CUDA_CHECK(hipFreeAsync(___));

  // Verify the data on the host is correct


  // Destroy the CUDA stream
  //CUDA_CHECK(hipStreamDestroy(stream));

  // If the program makes it this far, then the results are correct and
  // there are no run-time errors.  Good work!
  //std::cout << "Correct!" << std::endl;

  return 0;
}
