#include "hip/hip_runtime.h"
// C++ standard headers
#include <cassert>
#include <iostream>
#include <vector>

// CUDA headers
#include <hip/hip_runtime.h>

// local headers
#include "cuda_check.h"

// Here you can set the device ID that was assigned to you
#define MYDEVICE 0

__global__ void sum(const int* a, const int* b, int* c){
  c[threadIdx.x ]
}

///////////////////////////////////////////////////////////////////////////////
// Program main
///////////////////////////////////////////////////////////////////////////////
int main()
{
  // Choose one CUDA device
  CUDA_CHECK(hipSetDevice(MYDEVICE));

  // Create a CUDA stream to execute asynchronous operations on this device
  hipStream_t stream;
  CUDA_CHECK(hipStreamCreate(&stream));

  // Pointer and dimension for host memory
  size_t dimA = 8;
  std::vector<float> h_a(dimA);
  //std::vector<float> h_b(dimA);
  std::vector<float> h_c(dimA);

  // Allocate and initialize host memory
  for (uint i = 0; i < dimA; ++i) {
     h_a[i] = i;
     //h_b[i] = 2*i;
  }

  // Pointers for device memory
  float *d_a, *d_b, *d_c;

  // Part 1 of 5: allocate the device memory
  size_t memSize = dimA * sizeof(float);
  //size_t floatSize = sizeof(float);

  hipMalloc(&d_a, memSize);
  hipMalloc(&d_b, memSize);

  // CUDA_CHECK(hipMallocAsync(___));
  // CUDA_CHECK(hipMallocAsync(___));

  // Part 2 of 5: host to device memory copy
  // Hint: the raw pointer to the underlying array of a vector
  // can be obtained by calling std::vector<T>::data()
  hipMemcpy(d_a, h_a.data(), memSize, hipMemcpyHostToDevice);
  
  //CUDA_CHECK(hipMemcpyAsync(___));

  // Part 3 of 5: device to device memory copy
  //CUDA_CHECK(hipMemcpyAsync(___));
  hipMemcpy(d_b, d_a, memSize, hipMemcpyDeviceToDevice);

  // Clear the host memory
  //std::fill(h_a.begin(), h_a.end(), 0);

  // Part 4 of 5: device to host copy
  hipMemcpy(h_c.data(),d_c, memSize, hipMemcpyDeviceToHost);
  //CUDA_CHECK(hipMemcpyAsync(___));

  // Wait for all asynchronous operations to complete
  //CUDA_CHECK(hipStreamSynchronize(stream));
  sum<<<1, dimA, 0>>>(d_a,d_b,d_c);

  // Part 5 of 5: free the device memory
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);
  //CUDA_CHECK(hipFreeAsync(___));
  //CUDA_CHECK(hipFreeAsync(___));

  // Verify the data on the host is correct
  for (int i = 0; i < dimA; ++i) {
    std::cout << h_c[i] << '\n';
    //assert(h_a[i] == (float)i);
  }

  // Destroy the CUDA stream
  CUDA_CHECK(hipStreamDestroy(stream));

  // If the program makes it this far, then the results are correct and
  // there are no run-time errors.  Good work!
  std::cout << "Correct!" << std::endl;

  return 0;
}
